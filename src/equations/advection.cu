#include "hip/hip_runtime.h"
#include <iostream>
#include "equation.h"

Advection::Advection(double velocity) : _velocity(velocity) {}

Advection::Advection(json json_data) {
    _velocity = json_data.at("velocity");
}
__global__
void eval_advection_residual(double *phi, double *residual, double u, double dx, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        double phi_minus, phi_plus;
        if (i == 0) {
            phi_minus = phi[n-1];
            phi_plus = phi[i];
        }
        else {
            phi_minus = phi[i-1];
            phi_plus = phi[i];
        }
        residual[i] = -u * (phi_plus - phi_minus) / dx;
    }
}

void Advection::eval_residual(Field<double> &phi, Field<double> &residual, Domain &domain) {
    unsigned n_blocks = domain.number_blocks();
    unsigned block_size = domain.block_size();


    eval_advection_residual<<<n_blocks,block_size>>>(
        phi.data(), residual.data(), _velocity, domain.dx(), phi.length()
    );

    auto code = hipGetLastError();
    if (code != hipSuccess){
        std::cerr << "Cuda error in advection residual eval: " << hipGetErrorString(code) << std::endl;
        throw new std::runtime_error("Encountered cuda error");
    }
}
