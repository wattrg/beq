#include "hip/hip_runtime.h"
#include <exception>
#include <iostream>
#include <fstream>
#include <math.h>
#include <filesystem>
#include <stdexcept>

#include "config.h"

#define _STRINGIFY(x) #x
#define STRINGIFY(x) _STRINGIFY(x)

const int BLOCK_SIZE = 256;
int number_solutions = 0;

void print_header() {
    std::cout << "beq: Boltzmann equation solver\n";
    std::cout << "Git branch: " << STRINGIFY(GIT_BRANCH) << "\n";
    std::cout << "Git commit: " << STRINGIFY(GIT_HASH) << "\n";
    std::cout << "Build date: " << STRINGIFY(COMPILE_TIME) << "\n";
}

__global__
void eval_rhs(double *phi, double *residual, double u, double dx, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        double phi_minus, phi_plus;
        if (i == 0) {
            phi_minus = phi[n-1];
            phi_plus = phi[i];
        }
        else {
            phi_minus = phi[i-1];
            phi_plus = phi[i];
        }
        residual[i] = -u * (phi_plus - phi_minus) / dx;
    }
}

__global__
void apply_residual(double *phi, double *phi_new, double *residual, double dt, int n){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        phi_new[i] =  phi[i] + residual[i] * dt;
    }
}

void take_step(double *phi, double *phi_new, double *residual, double u, double dx, double dt, int n, int number_blocks) {
    eval_rhs<<<number_blocks, BLOCK_SIZE>>>(phi, residual, u, dx, n);
    apply_residual<<<number_blocks, BLOCK_SIZE>>>(phi, phi_new, residual, dt, n);

    // swap phi and phi_new
    double *phi_tmp = phi;
    phi = phi_new;
    phi_new = phi_tmp;
}

void read_initial_condition(double *phi, int n) {
    std::ifstream initial_condition("solution/phi_0.beq");
    std::string phi_ic;
    int i = 0;
    while (getline(initial_condition, phi_ic)) {
        if (i >= n) {
            initial_condition.close();
            throw new std::runtime_error("Too many values in IC");
        }
        phi[i] = std::stod(phi_ic); 
        i++;
    }
    initial_condition.close();
    if (i != n){
        throw new std::runtime_error("Too few values in IC");
    }
}

void write_solution(double *phi_gpu, double *phi_cpu, int n){
    // copy data to CPU
    hipMemcpy(phi_cpu, phi_gpu, n*sizeof(double), hipMemcpyDeviceToHost);

    // write contents of cpu buffer to file
    std::string file_name = "solution/phi_" + std::to_string(number_solutions) + ".beq";
    std::ofstream file(file_name);
    for (int i = 0; i < n; i++){
        file << phi_cpu[i];
        file << "\n";
    }
    file.close();
    number_solutions++;
}

int main() {
    print_header();


    // unpack some config data
    const Config config = Config("config/config.json");
    int N = config.number_cells;
    double L = config.length;
    double u = config.velocity;
    const int number_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    double dx = L / N;
    double dt = config.cfl * dx / u;
    int print_frequency = config.print_frequency;
    int plot_frequency = config.plot_frequency;
    double max_time = config.max_time;
    int max_step = config.max_step;

    // allocate memory on host
    double *phi = new double[N];
    read_initial_condition(phi, N);

    // allocate memory on GPU
    double *phi_gpu, *phi_new_gpu, *residual_gpu;
    hipMalloc(&phi_gpu, N*sizeof(double));
    hipMalloc(&phi_new_gpu, N*sizeof(double));
    hipMalloc(&residual_gpu, N*sizeof(double));


    // copy initial condition to the GPU
    hipMemcpy(phi_gpu, phi, N*sizeof(double), hipMemcpyHostToDevice);

    double t = 0;
    for (int step = 0; step < max_step; step++){
        take_step(phi_gpu, phi_gpu, residual_gpu, u, dx, dt, N, number_blocks);
        t += dt;

        if (step % plot_frequency == 0){
            write_solution(phi_gpu, phi, N);
        }

        if (step % print_frequency == 0){
            std::cout << "step: " << step << "\n";
        }

        if (t > max_time){
            std::cout << "Finished: reached max_time" << std::endl;
            break;
        }
    }
    write_solution(phi_gpu, phi, N);


    // free host memory
    delete [] phi;

    // free device memory
    hipFree(phi_gpu);
    hipFree(phi_new_gpu);
    hipFree(residual_gpu);
    
    return 0;
}
